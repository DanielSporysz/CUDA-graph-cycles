#include "hip/hip_runtime.h"
#pragma once
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>

#include <iostream>
#include <list>
#include <vector>

#include "../Graph IO Utilities/GraphReader.h"

#include"cudaAnalyzer.h"
#include "stack.h"
#include "PathsContainer.h"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__device__ void visitVertex(int toVisit, int destination, Stack* path, int* visitedVertices, PathsContainer* cycles, int* matrix, int count) {
	path->push(toVisit);
	visitedVertices[toVisit] = visited;

	for (int i = 0; i < count; i++)
	{
		if (matrix[toVisit * count + i] == connected) {
			if (i == destination) { // Found the destination
				path->push(destination);
				int* pathToAdd = path->makeCopy();
				cycles->addPath(pathToAdd, path->count);
				free(pathToAdd);
				path->pop();
			}
			else if (visitedVertices[i] == notVisited) { // Look futher
				visitVertex(i, destination, path, visitedVertices, cycles, matrix, count);
			}
		}
	}

	path->pop();
	visitedVertices[toVisit] = notVisited;
}

__global__ void beginVisting(PathsContainer* d_outputs, int* d_matrix, config_t d_config) {
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	int count = d_config.matrixSize;

	if (tid < d_config.matrixSize)
	{
		// PreAnalysis data preparations
		int* visitedVerticles = (int*)malloc(count * sizeof(int));
		for (int i = 0; i < count; i++) {
			visitedVerticles[i] = notVisited;
		}

		Stack path;
		new(&d_outputs[tid]) PathsContainer;

		// Analysis
		visitVertex(tid, tid, &path, visitedVerticles, &d_outputs[tid], d_matrix, count);

		free(visitedVerticles);
	}
}


__global__ void getOutputSize(PathsContainer* d_outputs, int* outputSize, config_t d_config) {
	*outputSize = 0;
	for (int i = 0; i < d_config.matrixSize; i++)
	{
		*outputSize += d_outputs[i].count;
	}
}

__global__ void transferOutputs(int* cycles, PathsContainer* d_outputs, config_t config) {
	int offset = 0;
	for (int i = 0; i < config.matrixSize; i++)
	{
		memcpy(&cycles[offset], d_outputs[i].paths, d_outputs[i].count * sizeof(*cycles));
		offset += d_outputs[i].count;
	}
}

__global__ void freePathsContainers(PathsContainer* d_outputs, int count) {
	for (int i = 0; i < count; i++)
	{
		if (&d_outputs[i] != NULL) {
			delete &d_outputs[i];
		}

	}
}

std::list<std::vector<int>> convertToList(int* mergedCycles, int count) {
	std::list<std::vector<int>> cycles;

	std::vector<int> *tmp = new std::vector<int>();
	int head = mergedCycles[0];
	tmp->push_back(head);

	for (int i = 1; i < count; i++)
	{
		tmp->push_back(mergedCycles[i]);

		if (head == mergedCycles[i]) {
			cycles.push_back(*tmp);
			tmp = new std::vector<int>();

			if (i + 1 < count) {
				head = mergedCycles[i + 1];
				tmp->push_back(head);
				i++;
			}
		}
	}

	return cycles;
}

std::list<std::vector<int>> findCycles(int* matrix, config_t config) {
	// Data preparation
	PathsContainer* d_outputs;
	int* d_matrix;
	int matrixSize = config.matrixSize * config.matrixSize * sizeof(*matrix);
	gpuErrchk(hipMalloc(&d_matrix, matrixSize));
	gpuErrchk(hipMemcpy(d_matrix, matrix, matrixSize, hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc(&d_outputs, config.matrixSize * sizeof(PathsContainer)));

	// Calculations
	beginVisting << <(config.matrixSize + 255) / 256, 256>> > (d_outputs, d_matrix, config);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	// Data size evaluation
	int* d_outputSize;
	gpuErrchk(hipMalloc(&d_outputSize, sizeof(int)));

	getOutputSize << <1, 1 >> > (d_outputs, d_outputSize, config);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	int* outputSize = (int*)malloc(sizeof(int));
	gpuErrchk(hipMemcpy(outputSize, d_outputSize, sizeof(*outputSize), hipMemcpyDeviceToHost));

	// Data transfer
	int *d_cycles;
	gpuErrchk(hipMalloc(&d_cycles, *outputSize * sizeof(*matrix)));
	transferOutputs << <1, 1 >> > (d_cycles, d_outputs, config);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	int* mergedCycles = (int*)malloc(*outputSize * sizeof(*matrix));
	gpuErrchk(hipMemcpy(mergedCycles, d_cycles, *outputSize * sizeof(*matrix), hipMemcpyDeviceToHost));

	// Conversion
	std::list<std::vector<int>> cycles;
	cycles = convertToList(mergedCycles, *outputSize);

	// Clean up
	gpuErrchk(hipFree(d_matrix));
	gpuErrchk(hipFree(d_cycles));
	freePathsContainers << <1, 1 >> > (d_outputs, config.matrixSize);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipFree(d_outputSize));
	gpuErrchk(hipFree(d_outputs));

	free(mergedCycles);
	free(outputSize);

	return cycles;
}