#include "hip/hip_runtime.h"
#pragma once
#include"cudaAnalyzer.h"
#include "hip/hip_runtime.h"
#include ""
#include "vector"
#include "list"
#include <iostream>
#include "stack.h"
#include "PathsContainer.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void searchFrom(int** d_cycles, int* d_sizes, int testSize) {
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (tid < testSize)
	{
		d_sizes[tid] = 3;

		d_cycles[tid] = (int*)malloc(3 * sizeof(int));
		d_cycles[tid][0] = tid + 10;
		d_cycles[tid][1] = tid + 100;
		d_cycles[tid][2] = tid + 1000;
	}
}

__global__ void transferData(int* d_to, int** d_from, int* d_sizes, int verticles) {
	int id = 0;
	for (int i = 0; i < verticles; i++)
	{
		for (int j = 0; j < d_sizes[i]; j++)
		{
			d_to[id++] = d_from[i][j];
		}
	}
}

std::list<std::vector<int>> findCycles(int* matrix, config_t config) {
	std::list<std::vector<int>> cycles;

	int testSize = 5;

	// Configuration
	dim3 block(64,64);
	dim3 grid;
	grid.x = (testSize + block.x - 1) / block.x;
	grid.y = 1;

	// Data preparation
	int** d_outputs;
	int* d_sizes;
	gpuErrchk(hipMalloc(&d_outputs, testSize * sizeof(int*)));
	gpuErrchk(hipMalloc(&d_sizes, testSize * sizeof(int)));
	gpuErrchk(hipMemset(d_sizes, 0, testSize * sizeof(int)));

	// Calculations
	searchFrom << <(testSize + 255) / 256, 256 >> > (d_outputs, d_sizes, testSize);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipPeekAtLastError());

	// Data size evaluation
	int* sizes = (int*)malloc(testSize * sizeof(int));
	gpuErrchk(hipMemcpy(sizes, d_sizes, testSize * sizeof(int), hipMemcpyDeviceToHost));
	int count = 0;
	for (int i = 0; i < testSize; i++)
	{
		count += sizes[i];
	}

	// Data transfer
	int* d_mergedOutputs;
	gpuErrchk(hipMalloc(&d_mergedOutputs, count * sizeof(int*)));
	transferData << <1, 1 >> > (d_mergedOutputs, d_outputs, d_sizes, testSize);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipPeekAtLastError());

	int* outputs = (int*)malloc(count * sizeof(int));
	gpuErrchk(hipMemcpy(outputs, d_mergedOutputs, count * sizeof(int*), hipMemcpyDeviceToHost));

	// DEBUG print
	std::cout << "Dynamically allocated memory transfer test: " << std::endl;
	for (int i = 0; i < count; i++)
	{
		std::cout << outputs[i] << " ";
	}
	std::cout << std::endl;

	return cycles; //TODO
}