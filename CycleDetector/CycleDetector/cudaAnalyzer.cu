#include "hip/hip_runtime.h"
#pragma once
#include"cudaAnalyzer.h"
#include "hip/hip_runtime.h"
#include ""
#include "vector"
#include "list"
#include <iostream>

__device__ int* d_array;

__global__ void searchFrom(int* adr, int testSize) {
	int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
	int threadPosInBlock = threadIdx.x +
		blockDim.x * threadIdx.y +
		blockDim.x * blockDim.y * threadIdx.z;
	int blockPosInGrid = blockIdx.x +
		gridDim.x * blockDim.y +
		gridDim.x * gridDim.y * blockIdx.z;
	int tid = blockPosInGrid * threadsPerBlock + threadPosInBlock;

	if (tid < testSize) {
		adr[tid] = 1;
	}
}

std::list<std::vector<int>> findCycles(int* matrix, config_t config) {
	std::list<std::vector<int>> cycles;

	dim3 block(8, 8, 8);
	dim3 grid(16,16);
	int testSize = 5;

	hipMalloc(&d_array, testSize * sizeof(int));
	int* d_adr;
	hipGetSymbolAddress((void**)&d_adr, d_array);

	searchFrom<<<block, grid>>>(d_adr, testSize);
	hipDeviceSynchronize();

	int* array = (int*)malloc(testSize * sizeof(int));
	hipMemcpyFromSymbol(array, HIP_SYMBOL(d_array), testSize*sizeof(int));

	for (int i = 0; i < testSize; i++)
	{
		std::cout << "Array at " << i << " is " << array[i] << std::endl;
	}

	return cycles;
}