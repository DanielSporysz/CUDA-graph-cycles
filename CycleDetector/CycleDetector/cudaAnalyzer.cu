#include "hip/hip_runtime.h"
#pragma once
#include"cudaAnalyzer.h"
#include "hip/hip_runtime.h"
#include ""
#include "vector"
#include "list"
#include <iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void searchFrom(int** d_cycles, int* d_sizes, int testSize) {
	/*int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
	int threadPosInBlock = threadIdx.x +
		blockDim.x * threadIdx.y +
		blockDim.x * blockDim.y * threadIdx.z;
	int blockPosInGrid = blockIdx.x +
		gridDim.x * blockDim.y +
		gridDim.x * gridDim.y * blockIdx.z;
	int tid = blockPosInGrid * threadsPerBlock + threadPosInBlock;

	int tid = blockIdx.x *blockDim.x + threadIdx.x;
	int z = 2;*/

	/*if (tid < testSize) {
		hipMalloc(&d_cycles[tid], (tid + 1 ) * sizeof(int));
		d_sizes[tid] = tid + 1;

		for (int i = 0; i < tid; i++)
		{
			d_cycles[tid][i] = i;
		}
	}*/

	d_sizes[1] = 3;
	d_cycles[1] = (int*)malloc(3 * sizeof(int));
	d_cycles[1][0] = 2;
	d_cycles[1][1] = 1;
	d_cycles[1][2] = 0;
}

__global__ void transferData(int** d_from, int* d_to, int* d_sizes, int verticles) {
	int id = 0;
	for (int i = 0; i < verticles; i++)
	{
		for (int j = 0; j < d_sizes[i]; j++)
		{
			d_to[id++] = d_from[i][j];
		}
	}
}

std::list<std::vector<int>> findCycles(int* matrix, config_t config) {
	std::list<std::vector<int>> cycles;

	// Configuration
	dim3 grid(1,1);
	dim3 block(1, 1, 1);

	int testSize = 5;

	// Data preparation
	int** d_outputs;
	int* d_sizes;
	gpuErrchk(hipMalloc(&d_outputs, testSize * sizeof(int*)));
	gpuErrchk(hipMalloc(&d_sizes, testSize * sizeof(int)));
	gpuErrchk(hipMemset(d_sizes, 0, testSize * sizeof(int)));

	// Calculations
	searchFrom<<<grid, block>>>(d_outputs, d_sizes, testSize);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	// Data size evaluation
	int* sizes = (int*)malloc(testSize * sizeof(int));
	gpuErrchk(hipMemcpy(sizes, d_sizes, testSize * sizeof(int), hipMemcpyDeviceToHost));
	int count = 0;
	for (int i = 0; i < testSize; i++)
	{
		count += sizes[i];
	}

	// Data transfer
	int* d_mergedOutputs;
	gpuErrchk(hipMalloc(&d_mergedOutputs, count * sizeof(int*)));
	transferData << <1, 1 >> > (d_outputs, d_mergedOutputs, d_sizes, testSize);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	int* outputs = (int*)malloc(testSize * sizeof(int));
	gpuErrchk(hipMemcpy(outputs, d_mergedOutputs, count * sizeof(int*), hipMemcpyDeviceToHost));

	// DEBUG print
	std::cout << "Dynamically allocated memory transfer test: " << std::endl;
	for (int i = 0; i < count; i++)
	{
		std::cout << outputs[i] << " ";
	}

	return cycles; //TODO
}